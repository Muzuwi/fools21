#include "hip/hip_runtime.h"
#include <cstdint>
#include <cassert>
#include <chrono>
#include <fmt/format.h>

typedef uint32_t u32;
typedef uint16_t u16;
typedef uint8_t u8;

struct ForcerContext {
	u8 rng_state[4];
	u8 work_buffer[8*8];
	u8 decompress_buffer[0x30];
	u8 preloaded_map[8*8*9];
	u8* found_byte;
};

static const unsigned seed_count = 0xfffffff + 1;
static const unsigned block_count = 32;
static const unsigned threads_per_block = 1024;
static const unsigned core_count = block_count * threads_per_block;
static const unsigned seeds_per_thread = seed_count / core_count;
static const unsigned context_size = sizeof(ForcerContext) * core_count;


__device__ u8 rng_next(ForcerContext* ctx) {
	if(!ctx) return 0x0;

	ctx->rng_state[0]++;
	ctx->rng_state[1] = ctx->rng_state[3] ^ ctx->rng_state[0] ^ ctx->rng_state[1];
	ctx->rng_state[2] = ctx->rng_state[1] + ctx->rng_state[2];
	ctx->rng_state[3] = ctx->rng_state[3] + ((ctx->rng_state[2]>>1) ^ ctx->rng_state[1]);

	return ctx->rng_state[3];
}

__device__ void rng_reinitialize(ForcerContext* ctx, u32 seed) {
	if(!ctx) return;

	ctx->rng_state[3] = seed & 0xffu;
	ctx->rng_state[2] = (seed >> 8u) & 0xffu;
	ctx->rng_state[1] = (seed >> 16u) & 0xffu;
	ctx->rng_state[0] = (seed >> 24u) & 0xffu;

	for(unsigned i = 0; i < 0x10; ++i)
		rng_next(ctx);
}

__device__ u8 gen_ctl(ForcerContext* ctx, u32 seed, u16 x, u16 y) {
	if(!ctx) return 0x0;

	u8 masked_x = x & 3,
	   masked_y = y & 3;

	u32 mask = ((u32)(y & 0xFFFC) << 16u) | (x & 0xFFFC);
	rng_reinitialize(ctx, seed ^ mask);
	u8 b1 = rng_next(ctx) & 7;

	static const u8 dd0c_lookup[128] = {
			0x05,0x0B,0x06,0x00,0x0E,0x05,0x0B,0x03,0x09,0x0E,0x00,0x00,0x00,0x0C,0x00,0x00,0x00,0x0C,0x05,0x06,0x06,0x0D,0x0A,0x09,0x09,0x0B,0x06,0x00,0x00,0x05,0x0A,0x00,0x05,0x0B,0x06,0x00,0x0B,0x06,0x09,0x07,0x00,0x09,0x07,0x0A,0x00,0x05,0x0A,0x00,0x00,0x09,0x06,0x00,0x03,0x06,0x0C,0x05,0x00,0x09,0x0F,0x0A,0x00,0x05,0x0A,0x00,0x05,0x0A,0x00,0x00,0x0F,0x06,0x05,0x03,0x09,0x0F,0x0A,0x00,0x00,0x0C,0x00,0x00,0x05,0x0B,0x03,0x06,0x0E,0x00,0x00,0x0D,0x09,0x06,0x00,0x0C,0x00,0x0D,0x03,0x0A,0x00,0x09,0x06,0x00,0x06,0x05,0x0A,0x05,0x09,0x0F,0x06,0x0C,0x00,0x0C,0x09,0x0A,0x00,0x0C,0x00,0x00,0x03,0x0F,0x07,0x03,0x00,0x0D,0x0A,0x00,0x00,0x0C,0x00,0x00
	};

	auto offset = 4 * masked_y + masked_x;
	auto table_offset = offset + (b1 << 4u);

	u8 lookup_value = dd0c_lookup[table_offset];
	u8 b2 = rng_next(ctx) & 0x30;
	b2 |= lookup_value;

	auto result = (((x&0xFF) | (y&0xFF)) & 0xFC) | ((x>>8u) | (y>>8u));
	if(result != 0)
		return b2;
	else
		return b2 & 0x0F;
}

__device__ void map_decompress(ForcerContext* ctx, u8 new_tile) {
	if(!ctx) return;

	//  Copy tiles to not trample over things while modifying
	memcpy(&ctx->decompress_buffer[0], &ctx->work_buffer[8], 0x30);

	for(unsigned i = 8; i < 8*7; ++i) {
		if(ctx->decompress_buffer[i - 8] != new_tile)
			continue;
		if(((i&7) == 0) || ((i&7) == 7))
			continue;

		u8 v = rng_next(ctx);
		if(v & 1) {
			ctx->work_buffer[i-1] = new_tile;
		}
		if(v & 2) {
			ctx->work_buffer[i+1] = new_tile;
		}
		if(v & 4) {
			ctx->work_buffer[i-8] = new_tile;
		}
		if(v & 8) {
			ctx->work_buffer[i+8] = new_tile;
		}
	}
}

__device__ void map_place_tile_prob(ForcerContext* ctx, u8 old, u8 new_tile, u8 threshold) {
	if(!ctx) return;

	for(unsigned i = 0; i < 8*8; ++i) {
		if(ctx->work_buffer[i] != old)
			continue;
		u8 v = rng_next(ctx);
		if(v >= threshold)
			continue;

		ctx->work_buffer[i] = new_tile;
	}
}

//  Same as above, but skips lines 0 and 7 and tiles 0,7 on each line
//  presumably to avoid softlocking
__device__ void map_place_tile_prob_safe(ForcerContext* ctx, u8 old, u8 new_tile, u8 threshold) {
	if(!ctx) return;

	for(unsigned i = 8; i < 8*7; ++i) {
		if((i&7) == 0 || (i&7) == 7)
			continue;
		if(ctx->work_buffer[i] != old)
			continue;
		u8 v = rng_next(ctx);
		if(v >= threshold)
			continue;

		ctx->work_buffer[i] = new_tile;
	}
}

__device__ void map_place_tile_at(ForcerContext* ctx, u8 tile, u8 x, u8 y) {
	x = x & 0x0f;
	y = y & 0x0f;
	ctx->work_buffer[y * 8 + x] = tile;
}

__device__ void map_place_line_impl(ForcerContext* ctx, u8 tile, u8 pos1, u8 pos2) {
	if(!ctx) return;

	u8 x1 = (pos1 >> 4u) & 0xf,
			x2 = (pos2 >> 4u) & 0xf,
			y1 = pos1 & 0x0f,
			y2 = pos2 & 0x0f;

	u8 ystep = (y1 < y2) ? 0x01 : 0xFF,
			xstep = (x1 < x2) ? 0x01 : 0xFF;
	while((x1 != x2) || (y1 != y2)) {
		map_place_tile_at(ctx, tile, x1, y1);
		if(x1 != x2)
			x1 += xstep;
		map_place_tile_at(ctx, tile, x1, y1);
		if(y1 != y2)
			y1 += ystep;
		map_place_tile_at(ctx, tile, x1, y1);
	}
}

__device__ void map_place_line(ForcerContext* ctx, u8 tile, u8 start, u8 end) {
	if(!ctx) return;

	u8 x = rng_next(ctx) & 7;
	while(x == 0 || x == 7)
		x = rng_next(ctx) & 7;
	u8 y = rng_next(ctx) & 7;
	while(y == 0 || y == 7)
		y = rng_next(ctx) & 7;

	u8 point_pos = (x << 4u) | y;

	map_place_line_impl(ctx, tile, start, point_pos);
	map_place_line_impl(ctx, tile, point_pos, end);
}


__device__ void maybe_map_gen(ForcerContext* ctx, u8 old, u8 new_tile, u8 threshold, u8 hfindpathflags, u8 hmultiplier, u8 hfindpathxprogress, u8 hmultiplybuffer) {
	if(!ctx) return;

	//  b - old tile
	//  c - new tile
	//  d - threshold

	//  ???? - findpathflags
	//  ???? - multiplier
	//  ???? - findpathxprogress
	//  ???? - multiplybuffer

	//  tile - hmutatewx
	unsigned i = 8, c = 0x30;
	while(c > 0) {
		if((i&7) == 0 || (i&7) == 7) {
			i++;
			c--;
			continue;
		}
		if(ctx->work_buffer[i] != old) {
			i++;
			c--;
			continue;
		}

		u8 v = rng_next(ctx);
		if(v < threshold) {
			i++;
			c--;
			continue;
		}

		i -= 0x8;
		if(hfindpathflags != 0) {
			if(ctx->work_buffer[i] != hfindpathflags) {
				i += 0x9;
				c--;
				continue;
			}
		}

		i += 0x10;
		if(hmultiplier != 0) {
			if(ctx->work_buffer[i] != hmultiplier) {
				i -= 0x7;
				c--;
				continue;
			}
		}

		i -= 0x09;
		if(hfindpathxprogress != 0) {
			if(ctx->work_buffer[i] != hfindpathxprogress) {
				i += 2;
				c--;
				continue;
			}
		}

		i += 2;
		if(hmultiplybuffer != 0) {
			if(ctx->work_buffer[i] != hmultiplybuffer) {
				c--;
				continue;
			}
		}

		ctx->work_buffer[i-1] = new_tile;
		c--;
	}
}


__device__ void gen_map_data(ForcerContext* ctx, u32 seed, u16 x, u16 y) {
	if(!ctx) return;
	//fmt::print("Generating map[{},{}] with seed={:08x}\n", x, y, seed);

	//  Generate control byte
	auto ctl = gen_ctl(ctx, seed, x,y);

	//  Fill work buffer with 0F
	memset(&ctx->work_buffer[0], 0x0F, 8 * 8);

	u8 var1 = 0;
	if(ctl & 1)
		var1 = 0x74;
	if(ctl & 2)
		var1 = 0x04;
	if(ctl & 4)
		var1 = 0x47;
	if(ctl & 8)
		var1 = 0x40;

	//  Reinitialize RNG
	u32 seed_mask = ((u32)x << 16u) | y;
	rng_reinitialize(ctx, seed ^ seed_mask);


	//  World gen?
	if(ctl & 1)
		map_place_line(ctx, 0xA, var1, 0x74);
	if(ctl & 2)
		map_place_line(ctx, 0xA, var1, 0x04);
	if(ctl & 4)
		map_place_line(ctx, 0xA, var1, 0x47);
	if(ctl & 8)
		map_place_line(ctx, 0xA, var1, 0x40);


	//  "Decompression"?
	map_decompress(ctx, 0x0A);


	//  Place exits
	if(ctl & 8) {
		map_place_tile_at(ctx, 0xa, 0x3, 0x0);
		map_place_tile_at(ctx, 0xa, 0x4, 0x0);
	}
	if(ctl & 4) {
		map_place_tile_at(ctx, 0xa, 0x3, 0x7);
		map_place_tile_at(ctx, 0xa, 0x4, 0x7);
	}
	if(ctl & 2) {
		map_place_tile_at(ctx, 0xa, 0x0, 0x3);
		map_place_tile_at(ctx, 0xa, 0x0, 0x4);
	}
	if(ctl & 1) {
		map_place_tile_at(ctx, 0xa, 0x7, 0x3);
		map_place_tile_at(ctx, 0xa, 0x7, 0x4);
	}

	//  Biome specific generation
	auto biome_ctl = (ctl >> 4u) & 0x3;
	switch(biome_ctl) {
		case 0: {
			map_place_tile_prob(ctx, 0xa, 0xb, 0x30);
			map_decompress(ctx, 0x0B);
			maybe_map_gen(ctx, 0x0f, 0x6c, 0x20, 0x0f, 0x0a, 0x0, 0x0);
			maybe_map_gen(ctx, 0x0f, 0x6f, 0x20, 0x0a, 0x0f, 0x0, 0x0);
			maybe_map_gen(ctx, 0x0f, 0x6e, 0x20, 0x0, 0x0, 0x0a, 0x0f);
			maybe_map_gen(ctx, 0x0f, 0x6d, 0x20, 0x0, 0x0, 0x0f, 0x0a);
			map_place_tile_prob(ctx, 0xa, 0x74, 0x30);
			map_place_tile_prob(ctx, 0xa, 0x7a, 0x30);
			map_place_tile_prob_safe(ctx, 0x6c, 0x33, 0x40);
			map_place_tile_prob_safe(ctx, 0x6d, 0x32, 0x40);
			map_place_tile_prob_safe(ctx, 0x6e, 0x60, 0x40);
			map_place_tile_prob_safe(ctx, 0x6f, 0x34, 0x40);

			break;
		}
		case 1: {
			map_place_tile_prob(ctx, 0xa, 0x7b, 0x40);
			map_place_tile_prob(ctx, 0xa, 0x7a, 0x30);
			map_place_tile_prob(ctx, 0xa, 0xb, 0xd0);
			map_place_tile_prob_safe(ctx, 0x0a, 0x08, 0x20);
			break;
		}
		case 2: {
			//  Screw this
			assert(false);
			break;
		}
		case 3: {
			//  Screw this
			assert(false);
			break;
		}
		default: break;
	}
}

__device__ void generate_visible_map(ForcerContext* ctx, u32 seed) {
	if(!ctx) return;

	auto get_base = [](u8 x, u8 y) {
		if(y == 0) {
			return 0x0 + x*8;
		} else if(y == 1) {
			return 0xC0 + x*8;
		} else {
			return 0x180 + x*8;
		}
	};

	for(unsigned x = 0; x < 3; ++x) {
		for(unsigned y = 0; y < 3; ++y) {
			gen_map_data(ctx, seed, x, y+1);
			auto base = get_base(x,y);

			//  Copy lines such that the lines in nearby chunks are sequential in memory
			for(unsigned i = 0; i < 8; ++i) memcpy(&ctx->preloaded_map[base + i * 0x18], &ctx->work_buffer[i * 8], 8);
		}
	}
}

__device__ bool search_sequence(ForcerContext* ctx, unsigned windowx, unsigned windowy) {
	if(!ctx) return false;

	if(windowx >= 24 || windowx + 5 >= 24 || windowy >= 24 || windowy + 5 >= 24)
		return false;

	const u8 sequence[25] = {
			0x0b,0x0b,0x0b,0x74,0x0a,
			0x0f,0x0b,0x0f,0x0a,0x0a,
			0x0f,0x0f,0x0a,0x0a,0x0b,
			0x0f,0x0b,0x0a,0x0a,0x0a,
			0x0b,0x0b,0x0a,0x0a,0x74
	};

	unsigned c = 0;
	for(unsigned y = windowy; y < windowy + 5; ++y) {
		for(unsigned x = windowx; x < windowx + 5; ++x) {
			auto addr = y * 24 + x;
			if(ctx->preloaded_map[addr] != sequence[c])
				return false;
			c++;
			if(c == 25)
				return true;
		}
	}

	return false;
}

__global__ void forcer_entrypoint(void* context_pool_base, u32 cycle_base) {
	if(!context_pool_base)
		return;

	auto threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	auto blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;
	auto threadsPerBlock  = blockDim.x * blockDim.y;
	auto thread_number = blockNumInGrid * threadsPerBlock + threadNumInBlock;
	auto* ctx = (ForcerContext*)((u8*)context_pool_base + thread_number * sizeof(ForcerContext));

	u32 input_seed = cycle_base + thread_number;
	auto seed = (input_seed << 4u) | 0x01u;
	generate_visible_map(ctx, seed);

	//  Search the possible window for the sequence
	for(unsigned x = 6; x <= 13; ++x) {
		for(unsigned y = 6; y <= 13; ++y) {
			bool res = search_sequence(ctx, x,y);
			if(res) {
				printf("[Thread %d] SEED=%08x Found pattern occurence!\n", thread_number, seed);
				return;
			}
		}
	}
}


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


int main() {
	fmt::print("Bruteforce using {} blocks, {} threads per block, total {} CUDA threads\n", block_count, threads_per_block, core_count);
	fmt::print("Seeds per CUDA thread: {}\n", seeds_per_thread);
	fmt::print("Forcer context size: {} bytes\n",  context_size);

	void* alloc_base;
	gpuErrchk(hipMalloc((void**)&alloc_base, context_size));

	unsigned rounds = seed_count / core_count;
	fmt::print("Rounds: {}\n", rounds);

	auto force_start = std::chrono::high_resolution_clock::now();
	auto start = force_start;
	for(unsigned i = 0; i < rounds; i++) {
		u32 seed_base = i * core_count;
		forcer_entrypoint<<<block_count, threads_per_block>>>(alloc_base, seed_base);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

		auto end = std::chrono::high_resolution_clock::now();
		auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
		auto eta = ((rounds - i) * duration) / 1000;
		auto eta_mins = eta / 60;
		auto eta_secs = eta % 60;
		if((i % 8) == 0) {
			const auto since_start = std::chrono::duration_cast<std::chrono::seconds>(end - force_start).count();
			const auto sps = (since_start == 0) ? 0 : ((i+1)*core_count) / since_start;
			fmt::print("Progress: round {}/{}, seeds: {:07x}x-{:07x}x [{}%], eta={}m:{}s, {} seeds/s\n", i, rounds, seed_base, (seed_base + core_count), 100.0 * seed_base / seed_count,
			  eta_mins, eta_secs, sps);
		}
		start = end;
	}

	hipFree(alloc_base);
	return 0;
}
